//IN THE NAME OF ALLAH

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>



void merge(int *, int , int , int);

void merge_sort(int * arr, int right, int left){
    if (left<right){
        int middle = left + (right - left) / 2;
        merge_sort(arr, left, middle);
        merge_sort(arr, middle + 1, right);

        merge(arr, left, middle, right);
    }
}


void merge(int * arr, int left, int middle, int right){
    int i, j, k;
    int n1 = middle - left + 1;
    int n2 = right - middle;
    int left_arr[n1];
    int right_arr[n2];

    for (i = 0; i < n1; i++){
        left_arr[i] = arr[left + i];
    }
    for (j = 0; j < n2; j++){
        right_arr[j] = arr[j + middle + 1];
    }

    i = 0;
    j = 0;
    k = 0;

    while (i < n1 && j < n2) { 
        if (left_arr[i] <= right_arr[j]) { 
            arr[k] = left_arr[i]; 
            i++; 
        } 
        else{ 
            arr[k] = right_arr[j]; 
            j++; 
        } 
        k++; 
    } 

    while (i < n1) { 
        arr[k] = left_arr[i]; 
        i++; 
        k++; 
    } 
    while (j < n2) { 
        arr[k] = right_arr[j]; 
        j++; 
        k++; 
    }
}


void print_array(int * A, int size) { 
    int i; 
    for (i=0; i < size; i++) 
        printf("%d ", A[i]); 
    printf("\n"); 
} 

int main() 
{ 
    int size = 0;
    scanf("%d", & size);
    int i = 0;
    int * arr = (int *) malloc(sizeof(int) * size);

    for (i = 0; i < size; i++){
        scanf("%d", &arr[i]);
    } 
    printf("Given array is: \n"); 
    print_array(arr, size); 
  
    merge_sort(arr, 0, size - 1); 
  
    printf("\nSorted array is: \n"); 
    print_array(arr, size); 
    return 0; 
} 